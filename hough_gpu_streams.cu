#include "hip/hip_runtime.h"
#include "opencv2/opencv.hpp"
#include "opencv2/imgproc/imgproc.hpp"
#include <opencv2/cudaimgproc.hpp>
#include <hip/hip_runtime.h>
#include <vector>
#include <cmath>
#include "omp.h"

struct line{
    int rho;
    float theta;
};

void saveAccumulatorImage(int* accu, int numangle, int numrho, const std::string& filename) {
    cv::Mat accuImage(numangle, numrho, CV_32SC1);
    for (int angle = 0; angle < numangle; ++angle) {
        for (int rho = 0; rho < numrho; ++rho) {
            int value = accu[(angle + 1) * (numrho + 2) + (rho + 1)];
            accuImage.at<int>(angle, rho) = value;
        }
    }

    cv::Mat normalized;
    cv::normalize(accuImage, normalized, 0, 255, cv::NORM_MINMAX, CV_8UC1);

    cv::imwrite("../results/"+filename, normalized);

    int scaleFactor = 7; 
    cv::Mat resized;
    cv::resize(normalized, resized, cv::Size(normalized.cols / scaleFactor, normalized.rows), cv::INTER_AREA);

    cv::imwrite("../results/accumulator_squeezed.png", resized);
}

__global__ void segment_image(const unsigned char* inputImage,
                                        unsigned char* subImage1,
                                        unsigned char* subImage2,
                                        unsigned char* subImage3,
                                        unsigned char* subImage4,
                                        int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= N || col >= N) return;

    int idx_in = row * N + col;

    int halfCols = N / 2;

    int out_row = row / 2;
    int out_col = col / 2;

    if ((row % 2 == 0) && (col % 2 == 0)) {
        subImage4[out_row * halfCols + out_col] = inputImage[idx_in];
    } else if ((row % 2 == 0) && (col % 2 == 1)) {
        subImage3[out_row * halfCols + out_col] = inputImage[idx_in];
    } else if ((row % 2 == 1) && (col % 2 == 0)) {
        subImage2[out_row * halfCols + out_col] = inputImage[idx_in];
    } else {
        subImage1[out_row * halfCols + out_col] = inputImage[idx_in];
    }
}

__global__ void extract_non_zero_coords(unsigned char* subImage,
                                        int* x_coords, int* y_coords, 
                                        int* count, int halfN, 
                                        int offsetX, int offsetY) {
    int y_idx = blockIdx.y * blockDim.y + threadIdx.y;
    int x_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if ( x_idx >= halfN || y_idx >= halfN) return;

    int idx = y_idx * halfN + x_idx;
    if (idx >= halfN * halfN) return;
    if (subImage[idx] != 0) {
        int global_x = x_idx * 2 + offsetX;
        int global_y = y_idx * 2 + offsetY;

        int index = atomicAdd(count, 1);
        x_coords[index] = global_x;
        y_coords[index] = global_y;
    }
}

__global__ void extract_non_zero(unsigned char* image,
                                 int * x, int *y,
                                 int *count, int N) {
    int x_coord = blockIdx.x * blockDim.x + threadIdx.x;
    int y_coord = blockIdx.y * blockDim.y + threadIdx.y;
    if (x_coord >= N || y_coord >= N) return;

    int idx = y_coord * N + x_coord;

    if (idx >= N * N) return;
    if (image[idx] != 0) {
        int index = atomicAdd(count, 1);
        x[index] = x_coord;
        y[index] = y_coord;    
    }
}

__global__ void sum_accumulators(int* output, int* accum1, int* accum2, int* accum3, int* accum4, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] +=  accum1[idx] + accum2[idx] + accum3[idx] + accum4[idx];
    }
}

__global__ void fill_trig_tables(float *sin_table, float *cos_table, float min_theta, float theta, int numangle, float irho){
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index < numangle){
        float angle = min_theta + index * theta;
        sin_table[index] = sinf(angle) *irho;
        cos_table[index] = cosf(angle) *irho;
    }
}

__global__ void fill_accum(int* accum,int* x_coords,int* y_coords,int num_points,float* cos_table,float* sin_table, int numangle,int numrho){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_points) return;

    int x = x_coords[idx];
    int y = y_coords[idx];

    for (int angle = 0; angle < numangle; angle++) {
        int r = roundf(x * cos_table[angle] + y * sin_table[angle]);
        r += (numrho - 1) / 2.f;
        atomicAdd(&accum[(angle + 1) * (numrho + 2) + r + 1], 1);
    }
}

__global__ void fill_accum_from_coords(int* accum,int* x_coords,int* y_coords,int *num_points,float* cos_table,float* sin_table, int numangle,int numrho){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= *num_points) return;

    int x = x_coords[idx];
    int y = y_coords[idx];

    for (int angle = 0; angle < numangle; angle++) {
        int r = roundf(x * cos_table[angle] + y * sin_table[angle]);
        r += (numrho - 1) / 2.f;
        atomicAdd(&accum[(angle + 1) * (numrho + 2) + r + 1], 1);
    }
}

__global__ void find_maxims(int* accum, int numangle, int numrho, int threshold, float min_theta, float theta_step, float rho_step, line* lines, int *current_size){
    int angle = blockDim.x * blockIdx.x + threadIdx.x;
    int rho = blockDim.y * blockIdx.y + threadIdx.y;

    if (angle < 1 || angle >= numangle) return;
    if (rho < 1 || rho >= numrho) return;

    int base = (angle+1) * (numrho+2) + rho + 1;
    if( accum[base] > threshold &&
        accum[base] > accum[base - 1] && accum[base] >= accum[base + 1] &&
        accum[base] > accum[base - numrho - 2] && accum[base] >= accum[base + numrho + 2]){
        int real_rho = (rho - (numrho - 1) *0.5f) * rho_step;
        float real_theta = min_theta + angle * theta_step;
        int index = atomicAdd(current_size,1);
        lines[index].rho = real_rho;
        lines[index].theta = real_theta;
    }
}

std::pair<int,line*> hough_parallel(unsigned char* d_img, int N, int threshold,
                                    float rho, float theta_step,double *duration,
                                    float min_theta=0.0, float max_theta=CV_PI){

    unsigned char *input;    

    hipMalloc(&input, N*N);
    hipMemcpy(input, d_img, N*N, hipMemcpyHostToDevice);

    auto start_count = omp_get_wtime();

    const float irho = 1.f / rho;
    const int max_rho = N + N;
    const int numangle = (int)cvFloor((max_theta - min_theta) / theta_step) + 1;
    const int numrho = cvRound(((max_rho*2+1))*irho);
    const int accu_size = (numangle+2)*(numrho+2);

    int *accum;
    float *sinvalues, *cosvalues;
    line *d_lines;
    int *d_counter, counter = 0;
    int *d_x_coords, *d_y_coords, *d_count;

    hipMalloc(&accum, accu_size * sizeof(int));
    hipMemset(accum, 0, accu_size * sizeof(int));

    hipMalloc(&sinvalues, numangle * sizeof(float));
    hipMalloc(&cosvalues, numangle * sizeof(float));
    hipMalloc(&d_lines, accu_size * sizeof(line));
    hipMalloc(&d_counter, sizeof(int));
    hipMemset(d_counter, 0, sizeof(int));
    hipMalloc(&d_x_coords, N * N * sizeof(int));
    hipMalloc(&d_y_coords, N * N * sizeof(int));
    hipMallocManaged(&d_count, sizeof(int));
    hipMemset(d_count, 0, sizeof(int));

    fill_trig_tables<<<1, numangle>>>(sinvalues, cosvalues, min_theta, theta_step, numangle, irho);

    dim3 blockDim(16, 16);
    dim3 gridDim((N + 15) / 16, (N + 15) / 16);
    extract_non_zero<<<gridDim, blockDim>>>(input, d_x_coords, d_y_coords, d_count, N);
    
    int h_count;
    hipMemcpy(&h_count, d_count, sizeof(int), hipMemcpyDeviceToHost);

    int block = 256;
    int grid = (h_count + block - 1) / block;
    fill_accum<<<grid, block>>>(accum, d_x_coords, d_y_coords, h_count, cosvalues, sinvalues, numangle, numrho);

    dim3 block_max(16, 16);
    dim3 grid_max((numangle+15)/16, (numrho+15)/16);
    find_maxims<<<grid_max, block_max>>>(accum, numangle, numrho, threshold, min_theta, theta_step, rho, d_lines, d_counter);

    hipMemcpy(&counter, d_counter, sizeof(int), hipMemcpyDeviceToHost);
    line* result = (line*)malloc(counter * sizeof(line));
    hipMemcpy(result, d_lines, counter * sizeof(line), hipMemcpyDeviceToHost);

    auto stop_count = omp_get_wtime();
    if (duration) *duration += stop_count - start_count;
    
    hipFree(input);
    hipFree(accum);
    hipFree(sinvalues);
    hipFree(cosvalues);
    hipFree(d_lines);
    hipFree(d_counter);
    hipFree(d_x_coords);
    hipFree(d_y_coords);
    hipFree(d_count);

    return {counter, result};
}

std::pair<int,line*> hough_parallel_segmented(unsigned char* d_img, int N, int threshold,
    float rho, float theta_step,double *duration,
    float min_theta=0.0, float max_theta=CV_PI) {

    unsigned char *input,*sub1, *sub2, *sub3, *sub4;    
    hipStream_t streams[4];
    for (int i = 0; i < 4; ++i) hipStreamCreate(&streams[i]);

    hipMalloc(&input, N*N);
    hipMemcpyAsync(input, d_img, N*N, hipMemcpyHostToDevice, streams[0]);

    auto start_count = omp_get_wtime();
    const float irho = 1.f / rho;
    
    const int height = N;
    const int width = N;
    const int halfN = N / 2;
    const int max_rho = width + height;

    const int numangle = (int)cvFloor((max_theta - min_theta) / theta_step) + 1;
    const int numrho = cvRound(((max_rho*2+1))*irho);
    const int accu_size = (numangle+2)*(numrho+2);

    int *acc1, *acc2, *acc3, *acc4, *accu_full;
    float *sinvalues, *cosvalues;
    line *d_lines;
    int *d_counter, counter = 0;

    hipMalloc(&sub1, halfN*halfN);
    hipMalloc(&sub2, halfN*halfN);
    hipMalloc(&sub3, halfN*halfN);
    hipMalloc(&sub4, halfN*halfN);

    hipMalloc(&acc1, accu_size * sizeof(int));
    hipMalloc(&acc2, accu_size * sizeof(int));
    hipMalloc(&acc3, accu_size * sizeof(int));
    hipMalloc(&acc4, accu_size * sizeof(int));
    hipMalloc(&accu_full, accu_size * sizeof(int));

    hipMalloc(&sinvalues, numangle * sizeof(float));
    hipMalloc(&cosvalues, numangle * sizeof(float));

    hipMalloc(&d_lines, accu_size * sizeof(line));
    hipMallocManaged(&d_counter, sizeof(int));

    hipMemsetAsync(acc1, 0, accu_size * sizeof(int), streams[0]);
    hipMemsetAsync(acc2, 0, accu_size * sizeof(int), streams[1]);
    hipMemsetAsync(acc3, 0, accu_size * sizeof(int), streams[2]);
    hipMemsetAsync(acc4, 0, accu_size * sizeof(int), streams[3]);
    hipMemsetAsync(accu_full, 0, accu_size * sizeof(int), streams[0]);
    hipMemsetAsync(d_counter, 0, sizeof(int), streams[0]);

    
    dim3 blockDim(16, 16);
    dim3 gridDim((N + 15) / 16, (N + 15) / 16);
    segment_image<<<gridDim, blockDim>>>(input, sub1, sub2, sub3, sub4, N);
 
    fill_trig_tables<<<1, numangle>>>(sinvalues, cosvalues, min_theta, theta_step, numangle, irho);

    int MPQ = 32;
    int total_points = halfN * halfN;
    int threads = (total_points + MPQ - 1) / MPQ;

    int max_points = halfN * halfN;
    int *x1, *y1, *x2, *y2, *x3, *y3, *x4, *y4;
    int *cnt1, *cnt2, *cnt3, *cnt4;
    
    hipMalloc(&x1, max_points * sizeof(int)); hipMalloc(&y1, max_points * sizeof(int)); hipMallocManaged(&cnt1, sizeof(int));
    hipMalloc(&x2, max_points * sizeof(int)); hipMalloc(&y2, max_points * sizeof(int)); hipMallocManaged(&cnt2, sizeof(int));
    hipMalloc(&x3, max_points * sizeof(int)); hipMalloc(&y3, max_points * sizeof(int)); hipMallocManaged(&cnt3, sizeof(int));
    hipMalloc(&x4, max_points * sizeof(int)); hipMalloc(&y4, max_points * sizeof(int)); hipMallocManaged(&cnt4, sizeof(int));

    hipMemsetAsync(cnt1, 0, sizeof(int), streams[0]);
    hipMemsetAsync(cnt2, 0, sizeof(int), streams[1]);
    hipMemsetAsync(cnt3, 0, sizeof(int), streams[2]);
    hipMemsetAsync(cnt4, 0, sizeof(int), streams[3]);

    extract_non_zero_coords<<<gridDim, blockDim,0, streams[0]>>>(sub1, x1, y1, cnt1, halfN, 1, 1);
    extract_non_zero_coords<<<gridDim, blockDim,0, streams[1]>>>(sub2, x2, y2, cnt2, halfN, 0, 1);
    extract_non_zero_coords<<<gridDim, blockDim,0, streams[2]>>>(sub3, x3, y3, cnt3, halfN, 1, 0);
    extract_non_zero_coords<<<gridDim, blockDim,0, streams[3]>>>(sub4, x4, y4, cnt4, halfN, 0, 0);

    for (int i = 0; i < 4; ++i) {
        hipStreamSynchronize(streams[i]);
    }

    fill_accum_from_coords<<<(*cnt1+255)/256, 256, 0, streams[0]>>>(acc1, x1, y1, cnt1, cosvalues, sinvalues, numangle, numrho);
    fill_accum_from_coords<<<(*cnt2+255)/256, 256, 0, streams[1]>>>(acc2, x2, y2, cnt2, cosvalues, sinvalues, numangle, numrho);
    fill_accum_from_coords<<<(*cnt3+255)/256, 256, 0, streams[2]>>>(acc3, x3, y3, cnt3, cosvalues, sinvalues, numangle, numrho);
    fill_accum_from_coords<<<(*cnt4+255)/256, 256, 0, streams[3]>>>(acc4, x4, y4, cnt4, cosvalues, sinvalues, numangle, numrho);

    for (int i = 0; i < 4; ++i) {
        hipStreamSynchronize(streams[i]);
    }

    sum_accumulators<<<(accu_size+255)/256, 256>>>(accu_full, acc1, acc2, acc3, acc4, accu_size);

    dim3 block_max(16, 16);
    dim3 grid_max((numangle+15)/16, (numrho+15)/16);

    hipDeviceSynchronize();

    find_maxims<<<grid_max, block_max>>>(accu_full, numangle, numrho, threshold, min_theta, theta_step, rho, d_lines, d_counter);

    
    hipMemcpy(&counter, d_counter, sizeof(int), hipMemcpyDeviceToHost);
    line* result = (line*)malloc(counter * sizeof(line));
    hipMemcpy(result, d_lines, counter * sizeof(line), hipMemcpyDeviceToHost);

    auto stop_count = omp_get_wtime();
    if (duration) *duration += stop_count - start_count;

    for (int i = 0; i < 4; ++i) hipStreamDestroy(streams[i]);
    hipFree(sub1); hipFree(sub2); hipFree(sub3); hipFree(sub4);
    hipFree(acc1); hipFree(acc2); hipFree(acc3); hipFree(acc4);
    hipFree(x1); hipFree(y1); hipFree(cnt1);
    hipFree(x2); hipFree(y2); hipFree(cnt2);
    hipFree(x3); hipFree(y3); hipFree(cnt3);
    hipFree(x4); hipFree(y4); hipFree(cnt4);
    hipFree(input);
    hipFree(sinvalues); hipFree(cosvalues);
    hipFree(d_lines); hipFree(d_counter);

    return {counter, result};
}

void drawHoughLines(const cv::Mat& houghLines, cv::Mat& outputImage, double scale = 1000.0, cv::Scalar color = cv::Scalar(0, 0, 255)) {
    for (int i = 0; i < houghLines.cols; ++i) {
        float rho   = houghLines.at<cv::Vec2f>(i)[0];
        float theta = houghLines.at<cv::Vec2f>(i)[1];
        double a = std::cos(theta), b = std::sin(theta);
        double x0 = a * rho, y0 = b * rho;
        cv::Point pt1(cvRound(x0 + scale * -b), cvRound(y0 + scale * a));
        cv::Point pt2(cvRound(x0 - scale * -b), cvRound(y0 - scale * a));
        cv::line(outputImage, pt1, pt2, color, 5, cv::LINE_AA);
    }
}

int main(int argc, char** argv){

    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if(deviceCount == 0){
        std::cout << "No CUDA device found" << std::endl;
        return -1;
    }

    int device = 0;
    hipGetDevice(&device);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    std::cout << "Cuda device: " << prop.name << std::endl;

    // check if image is provided
    if(argc < 4){
        std::cout << "not enough parametrs" << std::endl;
        return -1;
    }

    std::string filename = argv[1];
    int threshold = std::stoi(argv[2]);
    int n = std::stoi(argv[3]);

    std::string path ="../pictures/" + filename;
    double total_time_opencv_cpu = 0.0; 
    double total_time_opencv_gpu = 0.0;
    double total_time_opencv_gpu_full = 0.0;
    double total_time_mine_gpu_basic = 0.0;
    double total_time_mine_gpu_basic_full = 0.0;
    double total_time_mine_gpu_segmented = 0.0;
    double total_time_mine_gpu_segmented_full = 0.0;

    double total_lines_opencv_cpu = 0.0;
    double total_lines_opencv_gpu = 0.0;
    double total_lines_mine_gpu_basic = 0.0;
    double total_lines_mine_gpu_segmented = 0.0;

    for (int experiment = 0; experiment < n; ++experiment) {
        cv::Mat img = cv::imread(path, cv::IMREAD_COLOR);  
        if (img.empty()) {
            std::cerr << "Failed to load image: " << path << std::endl;
            return -1;
        }

        cv::Mat blur, edges;
        cv::Mat img_dst = img.clone();
        cv::Mat img_dst_mine = img.clone();
        cv::blur(img, blur, cv::Size(5, 5));
        cv::Canny(blur, edges, 50, 150, 3);

        if (n == 1)
            cv::imwrite("../results/edges.png", edges);
        int biggest = img.rows+img.rows;

        std::vector<cv::Vec2f> lines;
        auto start_cpu = omp_get_wtime();
        cv::HoughLines(edges, lines, 1, CV_PI/180, threshold);
        auto stop_cpu = omp_get_wtime();
        double duration_cpu = stop_cpu - start_cpu;
        total_time_opencv_cpu += duration_cpu;
        total_lines_opencv_cpu += lines.size();

        cv::cuda::GpuMat img_gpu, img_lines_gpu;
        cv::Mat img_lines_cpu;
        auto cv_hough = cv::cuda::createHoughLinesDetector(1, CV_PI/180, threshold);

        auto start_gpu_full = omp_get_wtime();
        img_gpu.upload(edges);
        auto start_gpu = omp_get_wtime();
        cv_hough->detect(img_gpu, img_lines_gpu);
        img_lines_gpu.download(img_lines_cpu);
        auto stop_gpu = omp_get_wtime();

        // Draw lines from img_lines_cpu
        if (n == 1)
            drawHoughLines(img_lines_cpu, img_dst, biggest); 

        double duration_gpu = stop_gpu - start_gpu;
        double duration_gpu_full = stop_gpu - start_gpu_full;
        total_time_opencv_gpu += duration_gpu;
        total_time_opencv_gpu_full += duration_gpu_full;
        total_lines_opencv_gpu += img_lines_gpu.cols;

        unsigned char *d_img = edges.ptr();
        int N = edges.rows;

        auto start_mine_basic = omp_get_wtime();
        std::pair<int,line*> result_basic = hough_parallel(d_img, N, threshold, 1, CV_PI/180,&total_time_mine_gpu_basic);
        auto stop_mine_basic = omp_get_wtime();
        total_time_mine_gpu_basic_full += (stop_mine_basic - start_mine_basic);
        total_lines_mine_gpu_basic += result_basic.first;
        if (n == 1){
            for (int i = 0; i < result_basic.first; ++i) {
                line line = result_basic.second[i];
                float theta = line.theta;
                float rho = line.rho;  
                double a = std::cos(theta);
                double b = std::sin(theta);
                double x0 = a * rho;
                double y0 = b * rho;
                cv::Point pt1(cvRound(x0 + biggest * (-b)), cvRound(y0 + biggest * (a)));
                cv::Point pt2(cvRound(x0 - biggest * (-b)), cvRound(y0 - biggest * (a)));
                cv::line(img_dst_mine, pt1, pt2, cv::Scalar(0, 0, 255), 5, cv::LINE_AA);
            } 
        }
        
        delete[] result_basic.second;

        auto start_mine_segmented = omp_get_wtime();
        std::pair<int,line*> result = hough_parallel_segmented(d_img, N, threshold, 1, CV_PI/180,&total_time_mine_gpu_segmented);
        auto stop_mine_segmented = omp_get_wtime();
        total_lines_mine_gpu_segmented += result.first;
        total_time_mine_gpu_segmented_full += (stop_mine_segmented - start_mine_segmented);
        

        delete[] result.second;
        std::string output_path_gpu = "../results/lines/gpu/";
        std::string output_path_cpu = "../results/lines/cpu/";

        std::string output_filename_gpu = "lines_gpu_opencv.png";
        std::string output_filename_mine_basic = "lines_mine_basic.png";

        cv::imwrite(output_path_gpu + output_filename_gpu, img_dst);
        cv::imwrite(output_path_gpu + output_filename_mine_basic, img_dst_mine);

        std::string output_filename_cpu = "lines_cpu_opencv.png";
        std::string output_filename_mine_segmented = "lines_mine_segmented.png"; 
        

    }

    std::cout << "\n=== AVERAGE TIMES OVER " << n << " EXPERIMENTS ===\n";
    std::cout << "OpenCV CPU Hough: " << (total_time_opencv_cpu / n)*1000.0 << " ms\n" << "Lines found: " << total_lines_opencv_cpu / n << "\n";
    std::cout << "OpenCV GPU Hough (kernel only): " << (total_time_opencv_gpu / n)*1000.0 << "ms\n" << "Lines found: " << total_lines_opencv_gpu / n << "\n";
    std::cout << "Mine GPU Hough (basic): " << (total_time_mine_gpu_basic / n)*1000.0 << "ms\n" << "Lines found: " << total_lines_mine_gpu_basic / n << "\n";
    std::cout << "Mine GPU Hough (segmented): " << (total_time_mine_gpu_segmented / n)*1000.0 << "ms\n" << "Lines found: " << total_lines_mine_gpu_segmented / n << "\n";
    std::cout << "OpenCV GPU Hough (full incl. transfer): " << (total_time_opencv_gpu_full / n)*1000.0 << "ms\n" << "Lines found: " << total_lines_opencv_gpu / n << "\n";
    std::cout << "Mine GPU Hough (basic incl. transfer): " << (total_time_mine_gpu_basic_full / n)*1000.0 << "ms\n" << "Lines found: " << total_lines_mine_gpu_basic / n << "\n";
    std::cout << "Mine GPU Hough (segmented incl. transfer): " << (total_time_mine_gpu_segmented_full / n)*1000.0 << "ms\n" << "Lines found: " << total_lines_mine_gpu_segmented / n << "\n";

    return 0;
}
