#include "hip/hip_runtime.h"
#include "opencv2/opencv.hpp"
#include "opencv2/imgproc/imgproc.hpp"
#include <opencv2/cudaimgproc.hpp>
#include <hip/hip_runtime.h>
#include <vector>
#include <cmath>
#include "omp.h"

struct line{
    int rho;
    float theta;
};

__global__ void segment_image(const unsigned char* inputImage,
                                        unsigned char* subImage1,
                                        unsigned char* subImage2,
                                        unsigned char* subImage3,
                                        unsigned char* subImage4,
                                        int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= N || col >= N) return;

    int idx_in = row * N + col;

    int halfCols = N / 2;

    int out_row = row / 2;
    int out_col = col / 2;

    if ((row % 2 == 0) && (col % 2 == 0)) {
        subImage4[out_row * halfCols + out_col] = inputImage[idx_in];
    } else if ((row % 2 == 0) && (col % 2 == 1)) {
        subImage3[out_row * halfCols + out_col] = inputImage[idx_in];
    } else if ((row % 2 == 1) && (col % 2 == 0)) {
        subImage2[out_row * halfCols + out_col] = inputImage[idx_in];
    } else {
        subImage1[out_row * halfCols + out_col] = inputImage[idx_in];
    }
}

__global__ void extract_non_zero_coords(unsigned char* subImage,
                                        int* x_coords, int* y_coords, 
                                        int* count, int halfN, 
                                        int offsetX, int offsetY) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= halfN || col >= halfN) return;

    int idx = row * halfN + col;
    if (subImage[idx] != 0) {
        int global_x = col * 2 + offsetX;
        int global_y = row * 2 + offsetY;

        int index = atomicAdd(count, 1);
        x_coords[index] = global_x;
        y_coords[index] = global_y;
    }
}

__global__ void extract_non_zero(unsigned char* image,
                                 int * x, int *y,
                                 int *count, int N) {
    int x_coord = blockIdx.x * blockDim.x + threadIdx.x;
    int y_coord = blockIdx.y * blockDim.y + threadIdx.y;
    if (x_coord >= N || y_coord >= N) return;

    int idx = y_coord * N + x_coord;

    if (idx >= N * N) return;
    if (image[idx] != 0) {
        int index = atomicAdd(count, 1);
        x[index] = x_coord;
        y[index] = y_coord;    
    }
}

__global__ void sum_accumulators(int* output, int* accum2, int* accum3, int* accum4, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] +=  accum2[idx] + accum3[idx] + accum4[idx];
    }
}

__global__ void fill_trig_tables(float *sin_table, float *cos_table, float min_theta, float theta, int numangle, float irho){
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index < numangle){
        float angle = min_theta + index * theta;
        sin_table[index] = sinf(angle) *irho;
        cos_table[index] = cosf(angle) *irho;
    }
}

__global__ void fill_accum(int* accum,int* x_coords,int* y_coords,int num_points,float* cos_table,float* sin_table, int numangle,int numrho){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_points) return;

    int x = x_coords[idx];
    int y = y_coords[idx];

    for (int angle = 0; angle < numangle; angle++) {
        int r = roundf(x * cos_table[angle] + y * sin_table[angle]);
        r += (numrho - 1) / 2.f;
        atomicAdd(&accum[(angle + 1) * (numrho + 2) + r + 1], 1);
    }
}

__global__ void fill_accum_from_coords(int* accum,int* x_coords,int* y_coords,int *num_points,float* cos_table,float* sin_table, int numangle,int numrho){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= *num_points) return;

    int x = x_coords[idx];
    int y = y_coords[idx];

    for (int angle = 0; angle < numangle; angle++) {
        int r = roundf(x * cos_table[angle] + y * sin_table[angle]);
        r += (numrho - 1) / 2.f;
        atomicAdd(&accum[(angle + 1) * (numrho + 2) + r + 1], 1);
    }
}

__global__ void find_maxims(int* accum, int numangle, int numrho, int threshold, float min_theta, float theta_step, float rho_step, line* lines, int *current_size){
    int angle = blockDim.x * blockIdx.x + threadIdx.x;
    int rho = blockDim.y * blockIdx.y + threadIdx.y;

    if (angle < 1 || angle >= numangle - 1 ) return;
    if (rho < 1 || rho >= numrho - 1 ) return;

    int base = (angle+1) * (numrho+2) + rho + 1;
    if( accum[base] > threshold &&
        accum[base] > accum[base - 1] && accum[base] >= accum[base + 1] &&
        accum[base] > accum[base - numrho - 2] && accum[base] >= accum[base + numrho + 2]){
        int real_rho = (rho - (numrho - 1) *0.5f) * rho_step;
        float real_theta = min_theta + angle * theta_step;
        int index = atomicAdd(current_size,1);
        lines[index].rho = real_rho;
        lines[index].theta = real_theta;
    }
}

std::pair<int,line*> hough_parallel(unsigned char* d_img, int N, int threshold,
                                    float rho, float theta_step,double *duration,
                                    float min_theta=0.0, float max_theta=CV_PI){

    unsigned char *input;    

    hipMalloc(&input, N*N);
    hipMemcpy(input, d_img, N*N, hipMemcpyHostToDevice);

    auto start_count = omp_get_wtime();

    const float irho = 1.f / rho;
    const int max_rho = N + N;
    const int numangle = (int)cvFloor((max_theta - min_theta) / theta_step) + 1;
    const int numrho = cvRound(((max_rho*2+1))*irho);
    const int accu_size = (numangle+2)*(numrho+2);

    int *accum;
    float *sinvalues, *cosvalues;
    line *d_lines;
    int *d_counter, counter = 0;
    int *d_x_coords, *d_y_coords, *d_count;

    hipMalloc(&accum, accu_size * sizeof(int));
    hipMemset(accum, 0, accu_size * sizeof(int));

    hipMalloc(&sinvalues, numangle * sizeof(float));
    hipMalloc(&cosvalues, numangle * sizeof(float));
    hipMalloc(&d_lines, accu_size * sizeof(line));
    hipMalloc(&d_counter, sizeof(int));
    hipMemset(d_counter, 0, sizeof(int));
    hipMalloc(&d_x_coords, N * N * sizeof(int));
    hipMalloc(&d_y_coords, N * N * sizeof(int));
    hipMallocManaged(&d_count, sizeof(int));
    hipMemset(d_count, 0, sizeof(int));

    fill_trig_tables<<<1, numangle>>>(sinvalues, cosvalues, min_theta, theta_step, numangle, irho);

    dim3 blockDim(16, 16);
    dim3 gridDim((N + 15) / 16, (N + 15) / 16);
    extract_non_zero<<<gridDim, blockDim>>>(input, d_x_coords, d_y_coords, d_count, N);
    
    int h_count;
    hipMemcpy(&h_count, d_count, sizeof(int), hipMemcpyDeviceToHost);

    int block = 256;
    int grid = (h_count + block - 1) / block;
    fill_accum<<<grid, block>>>(accum, d_x_coords, d_y_coords, h_count, cosvalues, sinvalues, numangle, numrho);

    dim3 block_max(16, 16);
    dim3 grid_max((numangle+15)/16, (numrho+15)/16);
    find_maxims<<<grid_max, block_max>>>(accum, numangle, numrho, threshold, min_theta, theta_step, rho, d_lines, d_counter);

    hipMemcpy(&counter, d_counter, sizeof(int), hipMemcpyDeviceToHost);
    line* result = (line*)malloc(counter * sizeof(line));
    hipMemcpy(result, d_lines, counter * sizeof(line), hipMemcpyDeviceToHost);

    auto stop_count = omp_get_wtime();
    if (duration) *duration += stop_count - start_count;

    hipFree(input);
    hipFree(accum);
    hipFree(sinvalues);
    hipFree(cosvalues);
    hipFree(d_lines);
    hipFree(d_counter);
    hipFree(d_x_coords);
    hipFree(d_y_coords);
    hipFree(d_count);

    
    return {counter, result};
}

std::pair<int,line*> hough_parallel_segmented(unsigned char* d_img, int N, int threshold,
    float rho, float theta_step,double *duration,
    float min_theta=0.0, float max_theta=CV_PI) {

    unsigned char *input,*sub1, *sub2, *sub3, *sub4;    
    hipStream_t streams[4];
    for (int i = 0; i < 4; ++i) hipStreamCreate(&streams[i]);

    hipMalloc(&input, N*N);
    hipMemcpyAsync(input, d_img, N*N, hipMemcpyHostToDevice, streams[0]);

    auto start_count = omp_get_wtime();
    const float irho = 1.f / rho;
    
    const int height = N;
    const int width = N;
    const int halfN = N / 2;
    const int max_rho = width + height;

    const int numangle = (int)cvFloor((max_theta - min_theta) / theta_step) + 1;
    const int numrho = cvRound(((max_rho*2+1))*irho);
    const int accu_size = (numangle+2)*(numrho+2);

    int *acc1, *acc2, *acc3, *acc4;
    float *sinvalues, *cosvalues;
    line *d_lines;
    int *d_counter, counter = 0;

    hipMalloc(&sub1, halfN*halfN);
    hipMalloc(&sub2, halfN*halfN);
    hipMalloc(&sub3, halfN*halfN);
    hipMalloc(&sub4, halfN*halfN);

    hipMalloc(&acc1, accu_size * sizeof(int));
    hipMalloc(&acc2, accu_size * sizeof(int));
    hipMalloc(&acc3, accu_size * sizeof(int));
    hipMalloc(&acc4, accu_size * sizeof(int));

    hipMalloc(&sinvalues, numangle * sizeof(float));
    hipMalloc(&cosvalues, numangle * sizeof(float));

    hipMalloc(&d_lines, accu_size * sizeof(line));
    hipMalloc(&d_counter, sizeof(int));

    hipMemsetAsync(acc1, 0, accu_size * sizeof(int), streams[0]);
    hipMemsetAsync(acc2, 0, accu_size * sizeof(int), streams[1]);
    hipMemsetAsync(acc3, 0, accu_size * sizeof(int), streams[2]);
    hipMemsetAsync(acc4, 0, accu_size * sizeof(int), streams[3]);
    hipMemsetAsync(d_counter, 0, sizeof(int), streams[0]);

    
    dim3 blockDim(16, 16);
    dim3 gridDim((N + 15) / 16, (N + 15) / 16);
    segment_image<<<gridDim, blockDim, 0, streams[0]>>>(input, sub1, sub2, sub3, sub4, N);
 
    fill_trig_tables<<<(numangle+255)/256, 256, 0, streams[0]>>>(sinvalues, cosvalues, min_theta, theta_step, numangle, irho);

    int MPQ = 32;
    int total_points = halfN * halfN;
    int threads = (total_points + MPQ - 1) / MPQ;

    int max_points = halfN * halfN;
    int *x1, *y1, *x2, *y2, *x3, *y3, *x4, *y4;
    int *cnt1, *cnt2, *cnt3, *cnt4;
    
    hipMalloc(&x1, max_points * sizeof(int)); hipMalloc(&y1, max_points * sizeof(int)); hipMalloc(&cnt1, sizeof(int));
    hipMalloc(&x2, max_points * sizeof(int)); hipMalloc(&y2, max_points * sizeof(int)); hipMalloc(&cnt2, sizeof(int));
    hipMalloc(&x3, max_points * sizeof(int)); hipMalloc(&y3, max_points * sizeof(int)); hipMalloc(&cnt3, sizeof(int));
    hipMalloc(&x4, max_points * sizeof(int)); hipMalloc(&y4, max_points * sizeof(int)); hipMalloc(&cnt4, sizeof(int));

    hipMemsetAsync(cnt1, 0, sizeof(int), streams[0]);
    hipMemsetAsync(cnt2, 0, sizeof(int), streams[1]);
    hipMemsetAsync(cnt3, 0, sizeof(int), streams[2]);
    hipMemsetAsync(cnt4, 0, sizeof(int), streams[3]);
    
    extract_non_zero_coords<<<gridDim, blockDim,0, streams[0]>>>(sub1, x1, y1, cnt1, halfN, 1, 1);
    extract_non_zero_coords<<<gridDim, blockDim,0, streams[1]>>>(sub2, x2, y2, cnt2, halfN, 0, 1);
    extract_non_zero_coords<<<gridDim, blockDim,0, streams[2]>>>(sub3, x3, y3, cnt3, halfN, 1, 0);
    extract_non_zero_coords<<<gridDim, blockDim,0, streams[3]>>>(sub4, x4, y4, cnt4, halfN, 0, 0);

    fill_accum_from_coords<<<(threads+255)/256, 256, 0, streams[0]>>>(acc1, x1, y1, cnt1, cosvalues, sinvalues, numangle, numrho);
    fill_accum_from_coords<<<(threads+255)/256, 256, 0, streams[1]>>>(acc2, x2, y2, cnt2, cosvalues, sinvalues, numangle, numrho);
    fill_accum_from_coords<<<(threads+255)/256, 256, 0, streams[2]>>>(acc3, x3, y3, cnt3, cosvalues, sinvalues, numangle, numrho);
    fill_accum_from_coords<<<(threads+255)/256, 256, 0, streams[3]>>>(acc4, x4, y4, cnt4, cosvalues, sinvalues, numangle, numrho);
    
    for (int i = 0; i < 4; ++i) hipStreamSynchronize(streams[i]);

    sum_accumulators<<<(accu_size+255)/256, 256, 0, streams[0]>>>(acc1, acc2, acc3, acc4, accu_size);

    dim3 block_max(16, 16);
    dim3 grid_max((numangle+15)/16, (numrho+15)/16);
    find_maxims<<<grid_max, block_max, 0, streams[0]>>>(acc1, numangle, numrho, threshold, min_theta, theta_step, rho, d_lines, d_counter);

    
    hipStreamSynchronize(streams[0]);
    
    hipMemcpy(&counter, d_counter, sizeof(int), hipMemcpyDeviceToHost);
    line* result = (line*)malloc(counter * sizeof(line));
    hipMemcpy(result, d_lines, counter * sizeof(line), hipMemcpyDeviceToHost);

    auto stop_count = omp_get_wtime();
    if (duration) *duration += stop_count - start_count;

    for (int i = 0; i < 4; ++i) hipStreamDestroy(streams[i]);
    hipFree(sub1); hipFree(sub2); hipFree(sub3); hipFree(sub4);
    hipFree(acc1); hipFree(acc2); hipFree(acc3); hipFree(acc4);
    hipFree(sinvalues); hipFree(cosvalues);
    hipFree(d_lines); hipFree(d_counter);

    return {counter, result};
}


int main(int argc, char** argv){

    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if(deviceCount == 0){
        std::cout << "No CUDA device found" << std::endl;
        return -1;
    }

    int device = 0;
    hipGetDevice(&device);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    std::cout << "Cuda device: " << prop.name << std::endl;

    // check if image is provided
    if(argc < 4){
        std::cout << "not enough parametrs" << std::endl;
        return -1;
    }

    std::string filename = argv[1];
    int threshold = std::stoi(argv[2]);
    int N = std::stoi(argv[3]);

    std::string path ="../pictures/" + filename;
    double total_time_opencv_cpu = 0.0; 
    double total_time_opencv_gpu = 0.0;
    double total_time_opencv_gpu_full = 0.0;
    double total_time_mine_gpu_basic = 0.0;
    double total_time_mine_gpu_basic_full = 0.0;
    double total_time_mine_gpu_segmented = 0.0;
    double total_time_mine_gpu_segmented_full = 0.0;

    double total_lines_opencv_cpu = 0.0;
    double total_lines_opencv_gpu = 0.0;
    double total_lines_mine_gpu_basic = 0.0;
    double total_lines_mine_gpu_segmented = 0.0;

    for (int experiment = 0; experiment < N; ++experiment) {
        cv::Mat img = cv::imread(path, cv::IMREAD_COLOR);  
        if (img.empty()) {
            std::cerr << "Failed to load image: " << path << std::endl;
            return -1;
        }

        cv::Mat blur, edges;
        cv::blur(img, blur, cv::Size(5, 5));
        cv::Canny(blur, edges, 50, 150, 3);
        int biggest = std::max(img.rows, img.cols);

        std::vector<cv::Vec2f> lines;
        auto start_cpu = omp_get_wtime();
        cv::HoughLines(edges, lines, 1, CV_PI/180, threshold);
        auto stop_cpu = omp_get_wtime();
        double duration_cpu = stop_cpu - start_cpu;
        total_time_opencv_cpu += duration_cpu;
        total_lines_opencv_cpu += lines.size();

        cv::cuda::GpuMat img_gpu, img_lines_gpu;
        cv::Mat img_lines_cpu;
        auto cv_hough = cv::cuda::createHoughLinesDetector(1, CV_PI/180, threshold);

        auto start_gpu_full = omp_get_wtime();
        img_gpu.upload(edges);
        auto start_gpu = omp_get_wtime();
        cv_hough->detect(img_gpu, img_lines_gpu);
        img_lines_gpu.download(img_lines_cpu);
        auto stop_gpu = omp_get_wtime();
        double duration_gpu = stop_gpu - start_gpu;
        double duration_gpu_full = stop_gpu - start_gpu_full;
        total_time_opencv_gpu += duration_gpu;
        total_time_opencv_gpu_full += duration_gpu_full;
        total_lines_opencv_gpu += img_lines_gpu.cols;

        unsigned char *d_img = edges.ptr();
        int N = edges.rows;

        auto start_mine_basic = omp_get_wtime();
        std::pair<int,line*> result_basic = hough_parallel(d_img, N, threshold, 1, CV_PI/180,&total_time_mine_gpu_basic);
        auto stop_mine_basic = omp_get_wtime();
        total_time_mine_gpu_basic_full += (stop_mine_basic - start_mine_basic);
        total_lines_mine_gpu_basic += result_basic.first;
        delete[] result_basic.second;

        auto start_mine_segmented = omp_get_wtime();
        std::pair<int,line*> result = hough_parallel_segmented(d_img, N, threshold, 1, CV_PI/180,&total_time_mine_gpu_segmented);
        auto stop_mine_segmented = omp_get_wtime();
        total_lines_mine_gpu_segmented += result.first;
        total_time_mine_gpu_segmented_full += (stop_mine_segmented - start_mine_segmented);
        delete[] result.second;
    }

    std::cout << "\n=== AVERAGE TIMES OVER " << N << " EXPERIMENTS ===\n";
    std::cout << "OpenCV CPU Hough: " << (total_time_opencv_cpu / N)*1000.0 << " ms\n" << "Lines found: " << total_lines_opencv_cpu / N << "\n";
    std::cout << "OpenCV GPU Hough (kernel only): " << (total_time_opencv_gpu / N)*1000.0 << "ms\n" << "Lines found: " << total_lines_opencv_gpu / N << "\n";
    std::cout << "Mine GPU Hough (basic): " << (total_time_mine_gpu_basic / N)*1000.0 << "ms\n" << "Lines found: " << total_lines_mine_gpu_basic / N << "\n";
    std::cout << "Mine GPU Hough (segmented): " << (total_time_mine_gpu_segmented / N)*1000.0 << "ms\n" << "Lines found: " << total_lines_mine_gpu_segmented / N << "\n";
    std::cout << "OpenCV GPU Hough (full incl. transfer): " << (total_time_opencv_gpu_full / N)*1000.0 << "ms\n" << "Lines found: " << total_lines_opencv_gpu / N << "\n";
    std::cout << "Mine GPU Hough (basic incl. transfer): " << (total_time_mine_gpu_basic_full / N)*1000.0 << "ms\n" << "Lines found: " << total_lines_mine_gpu_basic / N << "\n";
    std::cout << "Mine GPU Hough (segmented incl. transfer): " << (total_time_mine_gpu_segmented_full / N)*1000.0 << "ms\n" << "Lines found: " << total_lines_mine_gpu_segmented / N << "\n";

    return 0;
}
